#include "hip/hip_runtime.h"
//==============================================================================================
// Originally written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is
// distributed without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication
// along with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==============================================================================================

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>

#include <SFML/Graphics.hpp>
#include <chrono>
#include <iostream>
#include <mutex>
#include <thread>

#include "camera.h"
#include "color.h"
#include "cuda_utils.h"
#include "hittable_list.h"
#include "material.h"
#include "rtweekend.h"
#include "sphere.h"

std::mutex m;
int threadStarted = 0;

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__host__ __device__
    color
    ray_color(const ray &r, hittable &world, int depth) {
    hit_record rec;

    // If we've exceeded the ray bounce limit, no more light is gathered.
    if (depth <= 0)
        return color(0, 0, 0);

    if (world.hit(r, 0.001, infinity, rec)) {
        ray scattered;
        color attenuation;
        if (rec.mat_ptr->scatter(r, rec, attenuation, scattered))
            return attenuation * ray_color(scattered, world, depth - 1);
        return color(0, 0, 0);
    }

    vec3 unit_direction = unit_vector(r.direction());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

__host__
    hittable_list *
    random_scene() {
    hittable **objects_array;
    hipMallocManaged(&objects_array, sizeof(hittable *) * 5000);
    int idx = 0;

    auto ground_material = new lambertian(color(0.5, 0.5, 0.5));
    objects_array[idx++] = new sphere(point3(0, -1000, 0), 1000, ground_material);

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            auto choose_mat = random_double();
            point3 center(a + 0.9 * random_double(), 0.2, b + 0.9 * random_double());

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                material *sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color::random() * color::random();
                    sphere_material = new lambertian(albedo);
                    objects_array[idx++] = new sphere(center, 0.2, sphere_material);
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color::random(0.5, 1);
                    auto fuzz = random_double(0, 0.5);
                    sphere_material = new metal(albedo, fuzz);
                    objects_array[idx++] = new sphere(center, 0.2, sphere_material);
                } else {
                    // glass
                    sphere_material = new dielectric(1.5);
                    objects_array[idx++] = new sphere(center, 0.2, sphere_material);
                }
            }
        }
    }

    auto material1 = new dielectric(1.5);
    objects_array[idx++] = new sphere(point3(0, 1, 0), 1.0, material1);

    auto material2 = new lambertian(color(0.4, 0.2, 0.1));
    objects_array[idx++] = new sphere(point3(-4, 1, 0), 1.0, material2);

    auto material3 = new metal(color(0.7, 0.6, 0.5), 0.0);
    objects_array[idx++] = new sphere(point3(4, 1, 0), 1.0, material3);

    std::cerr << "Pushed " << idx << " objects" << std::endl;

    hittable_list *world_array = new hittable_list(objects_array, idx);

    return world_array;
}

// We can't generate the scene on host because of the use of virtual functions in the hittable objects.
// We need to do everything on the GPU
// This function takes a pointer to a world, and the GPU will create all objects in the scene
__global__ void random_scene_kernel(hittable_list **world, hittable **objects_array) {
    // hittable *objects_array[500];
    // hipMalloc(&objects_array, sizeof(hittable*) * 5000);
    //  this doesn't work for some reason???
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i == 0 && j == 0) {
        int idx = 0;

        auto ground_material = new lambertian(color(0.5, 0.5, 0.5));
        objects_array[idx++] = new sphere(point3(0, -1000, 0), 1000, ground_material);

        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                auto choose_mat = random_double();
                point3 center(a + 0.9 * random_double(), 0.2, b + 0.9 * random_double());

                if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                    material *sphere_material;

                    if (choose_mat < 0.8) {
                        // diffuse
                        auto albedo = color::random() * color::random();
                        sphere_material = new lambertian(albedo);
                        objects_array[idx++] = new sphere(center, 0.2, sphere_material);
                    } else if (choose_mat < 0.95) {
                        // metal
                        auto albedo = color::random(0.5, 1);
                        auto fuzz = random_double(0, 0.5);
                        sphere_material = new metal(albedo, fuzz);
                        objects_array[idx++] = new sphere(center, 0.2, sphere_material);
                    } else {
                        // glass
                        sphere_material = new dielectric(1.5);
                        objects_array[idx++] = new sphere(center, 0.2, sphere_material);
                    }
                }
            }
        }

        auto material1 = new dielectric(1.5);
        objects_array[idx++] = new sphere(point3(0, 1, 0), 1.0, material1);

        auto material2 = new lambertian(color(0.4, 0.2, 0.1));
        objects_array[idx++] = new sphere(point3(-4, 1, 0), 1.0, material2);

        auto material3 = new metal(color(0.7, 0.6, 0.5), 0.0);
        objects_array[idx++] = new sphere(point3(4, 1, 0), 1.0, material3);

        // std::cerr << "Pushed " << idx << " objects" << std::endl;

        *world = new hittable_list(objects_array, idx);
        printf("world ptr 1st obj %p\n", (*world)->objects_array[0]);
    }
}

class Tile {
   private:
    int id;
    double4 *pixel_array;
    std::thread thread;
    Tile **tiles;

   public:
    int x;
    int y;
    int tile_size;
    int samples;
    bool done = false;
    bool started = false;

    Tile(int id, Tile **tiles, int x, int y, int tile_size, int samples);
    ~Tile();
    void render(int image_width, int image_height, camera cam, hittable_list world, int max_depth);
    void renderThread(int image_width, int image_height, camera cam, hittable_list *world, int max_depth);
    double4 *getPixels();
};

Tile::Tile(int id, Tile **tiles, int x, int y, int tile_size, int samples) {
    this->id = id;
    this->tiles = tiles;
    this->x = x;
    this->y = y;
    this->tile_size = tile_size;
    this->samples = samples;
    this->pixel_array = (double4 *)malloc(tile_size * tile_size * sizeof(double4));
}

Tile::~Tile() {
    free(pixel_array);
}

void Tile::render(int image_width, int image_height, camera cam, hittable_list world, int max_depth) {
    m.lock();
    while (true) {
        if (threadStarted < 16 && ((tiles[id + 1] != NULL && tiles[id + 1]->started) || (x == 0 && y == 0))) {
            threadStarted++;
            m.unlock();
            break;
        } else {
            m.unlock();
            std::this_thread::sleep_for(std::chrono::milliseconds(20));
        }
    }

    started = true;
    for (int s = 0; s < samples; ++s) {
        for (int j = this->y; j < this->y + tile_size; ++j) {
            for (int i = this->x; i < this->x + tile_size; ++i) {
                int tile_x = i - this->x;
                int tile_y = j - this->y;
                int array_idx = tile_x + tile_y * tile_size;

                double4 pixel_color = this->pixel_array[array_idx];
                auto u = (i + random_double()) / (image_width - 1);
                auto v = (j + random_double()) / (image_height - 1);
                ray r = cam.get_ray(u, v);
                color c = ray_color(r, world, max_depth);
                double4 new_color = make_double4(c.x(), c.y(), c.z(), s);
                pixel_color = make_double4(c.x() + pixel_color.x, c.y() + pixel_color.y, c.z() + pixel_color.z, s);
                this->pixel_array[array_idx] = pixel_color;
            }
        }
    }

    this->done = true;
    m.lock();
    threadStarted--;
    m.unlock();
}

void Tile::renderThread(int image_width, int image_height, camera cam, hittable_list *world, int max_depth) {
    hittable_list temp = *world;
    std::thread thread(&Tile::render, this, image_width, image_height, cam, temp, max_depth);
    this->thread = std::move(thread);
}

double4 *Tile::getPixels() {
    return this->pixel_array;
}

__global__ void renderCuda(double4 *pixels, int image_width, int image_height, int samples, camera *cam, hittable_list **world_ptr, int max_depth) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    // const int id = i + j * blockIdx.y * blockDim.y;

    hittable_list temp_world = **world_ptr;

    if (i < image_width && j < image_height) {
        double4 pixel_color = make_double4(0, 0, 0, 0);
        for (int s = 1; s <= samples; ++s) {
            auto u = (i + random_double()) / (image_width - 1);
            auto v = (j + random_double()) / (image_height - 1);
            ray r = cam->get_ray(u, v);
            color c = ray_color(r, temp_world, max_depth);
            double4 new_color = make_double4(c.x(), c.y(), c.z(), s);
            pixel_color = make_double4(
                c.x() + pixel_color.x,
                c.y() + pixel_color.y,
                c.z() + pixel_color.z,
                s);
            pixels[i + j * image_width] = pixel_color;
        }
    }
}

void mergeTiles(sf::Uint8 *pixel_array, Tile **tiles, int nb_tiles, int tile_size, int image_height, int image_width, int samples) {
    for (int i = 0; i < nb_tiles; i++) {
        Tile *t = tiles[i];

        if (!t->started) continue;  // If the tile hasn't started rendering it's doesn't make sense to try to merge it

        int length = tile_size * tile_size;
        // Index of final pixel array (size of final image)
        int start_idx = t->x + (image_height - tile_size - t->y) * image_width;

        double4 *pixels = t->getPixels();

        for (int p = 0; p < length; p++) {
            // bottom pixel line should go on top
            int final_idx = (start_idx + p % tile_size) + (length - 1 - p) / tile_size * image_width;
            // Ditch pixels that were calculated but are out of the image or if they're black
            if (final_idx <= image_height * image_width && final_idx >= 0 && pixels[p].w > 0) {
                auto r = pixels[p].x;
                auto g = pixels[p].y;
                auto b = pixels[p].z;

                // Divide the color by the number of samples and gamma-correct for gamma=2.0.
                auto scale = 1.0 / pixels[p].w;
                r = sqrt(scale * r);
                g = sqrt(scale * g);
                b = sqrt(scale * b);

                pixel_array[final_idx * 4 + 0] = 256 * clamp(r, 0.0, 0.999);
                pixel_array[final_idx * 4 + 1] = 256 * clamp(g, 0.0, 0.999);
                pixel_array[final_idx * 4 + 2] = 256 * clamp(b, 0.0, 0.999);
                pixel_array[final_idx * 4 + 3] = 255u;
            }
        }
    }
}

void convertPixels(double4 *gpuPixels, sf::Uint8 *sfml_pixels, int image_width, int image_height) {
    for (int x = 0; x < image_width; x++) {
        for (int y = 0; y < image_height; y++) {
            int i = x + y * image_width;
            double4 pixel = gpuPixels[x + (image_height - y - 1) * image_width];

            auto scale = 1.0 / pixel.w;
            //printf("%f\n", pixel.w);
            auto r = sqrt(scale * pixel.x);
            auto g = sqrt(scale * pixel.y);
            auto b = sqrt(scale * pixel.z);
            sfml_pixels[i * 4 + 0] = 256 * clamp(r, 0.0, 0.999);
            sfml_pixels[i * 4 + 1] = 256 * clamp(g, 0.0, 0.999);
            sfml_pixels[i * 4 + 2] = 256 * clamp(b, 0.0, 0.999);
            sfml_pixels[i * 4 + 3] = 255u;
        }
    }
}

__host__ int main() {
    srand(time(NULL));

    // Image
    // const auto aspect_ratio = 16.f / 9.f;
    const auto aspect_ratio = 1;
    const int image_width = 640;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    const int samples_per_pixel = 20;  // 3 samples is the minimum to have a correct contrast / colors
    const int max_depth = 10;
    const int tile_size = 32;

    // World
    // auto world = random_scene();

    // Camera
    point3 lookfrom(13, 2, 3);
    point3 lookat(0, 0, 0);
    vec3 vup(0, 1, 0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;

    camera cam(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus);

    // Window setup
    sf::RenderWindow window(sf::VideoMode(image_width, image_height), "Ray Tracing with CUDA",
                            sf::Style::Titlebar | sf::Style::Close);
    auto desktop = sf::VideoMode::getDesktopMode();
    window.setPosition(sf::Vector2i(desktop.width / 2 - window.getSize().x / 2, desktop.height / 2 - window.getSize().y / 2));
    sf::Texture tex;
    sf::Sprite sprite;

    if (!tex.create(image_width, image_height)) {
        std::cerr << "Couldn't create texture!" << std::endl;
        return 1;
    }

    tex.setSmooth(false);

    sprite.setTexture(tex);

    // Render
    int total_tiles_x = (int)std::ceil((double)image_width / (double)tile_size);
    int total_tiles_x_pixels = total_tiles_x * tile_size;
    int total_tiles_y = (int)std::ceil((double)image_height / (double)tile_size);
    int total_tiles_y_pixels = total_tiles_y * tile_size;
    int nb_tiles = total_tiles_x * total_tiles_y;
    std::cerr << "Image size: " << image_width << "x" << image_height << "\n";
    std::cerr << "total_tiles_x: " << total_tiles_x << "\n";
    std::cerr << "total_tiles_y: " << total_tiles_y << "\n";
    std::cerr << "Nb tile: " << nb_tiles << "\n";
    std::cerr << "Sample size: " << samples_per_pixel << "\n";
    Tile **tiles = (Tile **)malloc(nb_tiles * sizeof(Tile *));

    double iStart = cpuSecond();

    // Increase the stack size
    // Can be removed if we remove recursions in kernels
    size_t stackSize;
    hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    std::cerr << "GPU Stack size: " << stackSize << std::endl;
    hipDeviceSetLimit(hipLimitStackSize, 65536);
    hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    std::cerr << "GPU Stack size: " << stackSize << std::endl;

    int grid_height = 32;
    int grid_width = grid_height;
    int grid_x = ceil(image_width / (double)grid_width) + 1;
    int grid_y = ceil(image_height / (double)grid_height) + 1;

    size_t array_size = grid_x * grid_y * grid_width * grid_height * sizeof(double4);

    printf("grid_x %d / grid_y %d\n", grid_x, grid_y);

    double4 *pixels;
    hipMallocManaged(&pixels, array_size);

    camera *dev_cam;
    printf("hipMalloc %d\n", hipMalloc(&dev_cam, sizeof(cam)));
    printf("hipMemcpy %d\n", hipMemcpy(dev_cam, &cam, sizeof(cam), hipMemcpyHostToDevice));

    hiprandState *curandStates = NULL;
    hipMalloc(&curandStates, grid_x * grid_y * grid_width * grid_height * sizeof(hiprandState));
    init_random_cuda<<<1, 1>>>(curandStates, rand());

    printf("%d\n", hipDeviceSynchronize());

    hittable_list **world;
    hipMalloc(&world, sizeof(hittable_list *));

    hittable **objects_array;
    hipMalloc(&objects_array, 1000 * sizeof(hittable *));

    random_scene_kernel<<<1, 1>>>(world, objects_array);

    printf("device sync %d\n", hipDeviceSynchronize());
    // renderCuda<<<dim3(grid_x, grid_y), dim3(grid_width, grid_height)>>>(pixels, image_width, image_height, samples_per_pixel, dev_cam, dev_world, max_depth);
    renderCuda<<<dim3(grid_x, grid_y), dim3(grid_width, grid_height)>>>(pixels, image_width, image_height, samples_per_pixel, dev_cam, world, max_depth);

    printf("device sync %d\n", hipDeviceSynchronize());

    // Render each tile
    // m.lock();
    // for (int i = nb_tiles - 1; i >= 0; i--) {
    //     int pixel_x = (i * tile_size) % total_tiles_x_pixels;
    //     int pixel_y = (i / (total_tiles_x)) * tile_size;
    //     Tile *tile = new Tile(nb_tiles - 1 - i, tiles, pixel_x, pixel_y, tile_size, samples_per_pixel);
    //     tiles[nb_tiles - 1 - i] = tile;
    //     tile->renderThread(total_tiles_x_pixels, total_tiles_y_pixels, cam, *world, max_depth);
    // }
    // m.unlock();

    sf::Uint8 *pixel_array_sfml = (sf::Uint8 *)malloc(image_width * image_height * 4 * sizeof(sf::Uint8));
    memset(pixel_array_sfml, 0, image_width * image_height * 4 * sizeof(sf::Uint8));
    tex.update(pixel_array_sfml);

    bool renderFinished = false;

    while (window.isOpen()) {
        sf::Event event;

        if (!renderFinished) {
            while (window.pollEvent(event)) {
                if (event.type == sf::Event::Closed) window.close();
            }

            // mergeTiles(pixel_array_sfml, tiles, nb_tiles, tile_size, image_height, image_width, samples_per_pixel);

            convertPixels(pixels, pixel_array_sfml, image_width, image_height);

            tex.update(pixel_array_sfml);

            window.clear();
            window.draw(sprite);
            window.display();

            // Sleep to not update too often
            // 10% performance hit with 720p 5 samples
            sf::sleep(sf::milliseconds(100));

            // bool tempFinished = true;
            // for (int i = 0; i < nb_tiles; i++) {
            //     Tile *t = tiles[i];
            //     if (!t->done) {
            //         tempFinished = false;
            //         break;
            //     }
            // }

            // if (tempFinished) {
            //     double iElaps = cpuSecond() - iStart;

            //     std::cerr << "Done render in " << iElaps << " seconds\n";

            //     // Merge one last time to be sure that the image is complete
            //     mergeTiles(pixel_array_sfml, tiles, nb_tiles, tile_size, image_height, image_width, samples_per_pixel);
            //     tex.update(pixel_array_sfml);

            //     window.clear();
            //     window.draw(sprite);
            //     window.display();

            //     tex.copyToImage().saveToFile("render.png");

            //     renderFinished = true;
            // }
        } else {
            window.waitEvent(event);
            if (event.type == sf::Event::Closed) window.close();
        }
    }
}
